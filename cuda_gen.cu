#include "hip/hip_runtime.h"
#pragma once

#include <cassert>
#include <limits>
#include <random>
#include <hiprand.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>

//Sample number with geometric distribution:
template <typename T>
struct geometric_distribution : public thrust::unary_function<T,T>
{
  const double p;
  const double maxvalinv;

  geometric_distribution(const double _p, const double _maxvalinv)
    : p(_p),
    maxvalinv(_maxvalinv)
  { }

  __device__
  T operator()(T x)
  {
    return __log2f(1.0f-maxvalinv*x)/__log2f(1.0f-p)+1;
    //return log1p(-_maxvalinv*(double)x)/log1p(-_p);
  }
};

//initialize prng only once (seed can be set nevertheless)
template <typename T>
class cuda_generator
{
  private:
    static cuda_generator<T> *_instance;
    thrust::device_vector<T> device_data;
    size_t N;
    hiprandGenerator_t prng;

    cuda_generator()
    {
      N = 0;
      //hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
      hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_MTGP32);
      //hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_MT19937);
    }

    void call_curand_generate()
    {
      hiprandGenerate(prng, thrust::raw_pointer_cast(device_data.data()), N);
    }

  public:
  static cuda_generator<T> *instance()
  {
    if(!_instance)
      _instance = new cuda_generator<T>();
    return _instance;
  }

  template <typename It>
  void generate_block(It dest, size_t size, double p,
                      unsigned long long seed = 0)
  {
    using value_type = typename std::iterator_traits<It>::value_type;			
    assert(p > 0 && p < 1);

    if (seed == 0) {
      seed = std::random_device{}();
    }

    //allocate memory on GPU if blocksize changed
    if(N != size)
    {
      device_data = thrust::device_vector<T>(size);
      N = size;
    }

    //set seed of prng
    hiprandSetPseudoRandomGeneratorSeed(prng, seed);

    //fill array with random numbers (should depend on size of the data type)
    call_curand_generate();

    fprintf(stderr, ".");

    //geometric distribution and prefix sum
    auto tbegin = thrust::make_transform_iterator(device_data.begin(), geometric_distribution<T>(p, 1.0/(std::numeric_limits<T>::max())));
    auto tend = thrust::make_transform_iterator(device_data.end(), geometric_distribution<T>(p, 1.0/(std::numeric_limits<T>::max())));
    thrust::inclusive_scan(tbegin, tend, device_data.begin());

    //read back data
    hipMemcpy(&dest[0], device_data.data().get(), N*sizeof(T), hipMemcpyDeviceToHost);
  }
};
template <typename T> cuda_generator<T> *cuda_generator<T>::_instance = 0;
/*
//TODO: not working!!! hiprand complains about not using a 64-bit prng.
template <>
void cuda_generator<unsigned long long>::call_curand_generate()
{
  hiprandGenerateLongLong(prng, thrust::raw_pointer_cast(device_data.data()), N);
}
*/

struct cuda_gen {
  template <typename It>
  static void generate_block(It dest, size_t size, double p,
                             unsigned int seed = 0)
  {
    using value_type = typename std::iterator_traits<It>::value_type;
    cuda_generator<value_type> *generator = cuda_generator<value_type>::instance();
    generator->generate_block(dest, size, p, seed);
  }

  template <typename It>
  static void generate_block(It begin, It end, double p,
                             unsigned int seed = 0)
  {
    generate_block(begin, end-begin, p, seed);
  }
};
