#include "hip/hip_runtime.h"
#include <cassert>
#include <limits>
#include <random>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#ifdef PROFILING
#include <hip/hip_runtime_api.h>
#endif
#include "util.h"
#include "timer.h"
#include <stocc/stocc.h>
#include <randomc/randomc.h>

#include <thrust/copy.h>
#include <thrust/find.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

#include <include/arg_parser.h>

#include <algorithm>
#include <cmath>
#include <utility>
#include <set>

#ifndef M_PI
#define M_PI 3.14159265358979323814
#endif

//initialize hiprand states
__global__ void initCurand(hiprandState *state, unsigned long long *seed)
{
  size_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(seed[threadId], 0, 0, &state[threadId]);
}

//sample random numbers with geometric distribution
template <typename T>
__global__ void sample(hiprandState *state, T *data, const size_t count, const T num, const double oneminusp)
{
  size_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
  size_t idx = threadId * num;
  hiprandState mState = state[threadId];
  //each thread generates 'num' random numbers with geometric distribution
  float lpinv = 1/log2(oneminusp);
  for(unsigned int i = 0; i < num && idx < count; i+=4, idx+=4)
  {
    float4 r = make_float4(__log2f(1-hiprand_uniform(&mState)),__log2f(1-hiprand_uniform(&mState)),__log2f(1-hiprand_uniform(&mState)),__log2f(1-hiprand_uniform(&mState)));
    if(i < num && idx < count)
      data[idx] = r.x*lpinv+1;
    if(i+1 < num && idx+1 < count)
      data[idx+1] = r.y*lpinv+1;
    if(i+2 < num && idx+2 < count)
      data[idx+2] = r.z*lpinv+1;
    if(i+3 < num && idx+3 < count)
      data[idx+3] = r.w*lpinv+1;
  }
  state[threadId] = mState;
}

//set elements to MAX to mark for removal
template <typename T>
__global__ void markRemove(T *data, const size_t count, T *remove, const size_t remove_count)
{
  size_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadId < remove_count)
    data[remove[threadId]] = T(-1);
}

//XXX Not working in all cases
#if 0
//replace k elements with k last ones
template <typename T>
__global__ void replace(T *data, const size_t count, T *remove, const size_t remove_count)
{
  size_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
  T idx = (T)-1;
  if(threadId < remove_count)
    idx = remove[threadId];

  if(idx != (T)-1 && data[idx] == (T)-1 && data[count-threadId-1] != (T)-1)
  {
    data[idx] = data[count-threadId-1];
    remove[threadId] = (T)-1;
  }
  if(idx == count-threadId-1)
    remove[threadId] = (T)-1;
  //otherwise replace in next iteration
}
#endif

//functor for filtering elements less than a constant
template <typename T>
struct less_functor
{
  T max_idx;
  less_functor(const T _max_idx)
    : max_idx(_max_idx)
  { }

  __host__ __device__
    bool operator()(T x)
    {
      return x < max_idx;
    }
};

//initialize prng only once (seed can be set nevertheless)
template <typename T>
class cuda_generator
{
  private:
    static cuda_generator<T> *_instance;

    T *device_data;

    //(GPU-)states for random number generators
    hiprandState *state;
    //seeds on CPU and GPU for each thread of the random number generator
    unsigned long long *seeds, *seeds_device;
    size_t N;
    const T num_threads = 1<<16;
    const T num_threads_per_group = 1<<3;
    const T num_groups = num_threads / num_threads_per_group;

    cuda_generator()
    {
      N = 0;
      hipMalloc((void**)&state, num_threads * sizeof(hiprandState));
      hipMalloc((void**)&seeds_device, num_threads * sizeof(unsigned long long));
      seeds = new unsigned long long[num_threads];
      device_data = 0;
    }

    ~cuda_generator()
    {
      hipFree((void**)&state);
      hipFree((void**)&seeds_device);
      delete [] seeds;
    }


  public:
    static cuda_generator<T> *instance()
    {
      if(!_instance)
        _instance = new cuda_generator<T>();
      return _instance;
    }

    template <typename It>
    void generate_block(It dest, size_t size, size_t k, size_t universe, double p,
      unsigned long long seed = 0)
    {
      using value_type = typename std::iterator_traits<It>::value_type;
      assert(p > 0 && p < 1);

      if (seed == 0)
      {
        seed = std::random_device{}();
      }

      //allocate memory on GPU if blocksize changed
      if(N != size)
      {
        if(device_data)
          hipFree(device_data);
        hipMalloc((void**)&device_data, sizeof(T)*size);
        N = size;
      }
      for(int i = 0; i < num_threads; i++)
        seeds[i] = seed+i;
      hipMemcpy(seeds_device, seeds, num_threads*sizeof(unsigned long long), hipMemcpyHostToDevice);
      //initialize prng-states
      initCurand<<<num_groups, num_threads_per_group>>>(state, seeds_device);
      //fill array with geometrically distributed random numbers
      T samples_per_thread = max((T)ceil(N/(double)num_threads), (T)1);
      T num_indices = 0;
      timer t;
      do {
        sample<<<num_groups, num_threads_per_group>>>(state, device_data, N, samples_per_thread, 1.0-p);
        //prefix-sum for calculating array-indices
        thrust::inclusive_scan(thrust::device, device_data, device_data+N, device_data);
        //count_if is faster than the other two.
        num_indices = thrust::count_if(thrust::device, device_data, device_data+N, less_functor<T>((T)universe));
        //num_indices = thrust::find_if(thrust::device, device_data, device_data+N, less_functor<T>((T)universe)) - device_data;
        //num_indices = thrust::copy_if(thrust::device, device_data, device_data+N, device_data, less_functor<T>((T)universe)) - device_data;
      } while(num_indices < k);

      //sample num_indices - k elements to remove
      std::mt19937 gen(seed);
      std::uniform_int_distribution<T> dist(0, num_indices-1);
      std::set<T> remove;

      while(remove.size() < num_indices - k)
        remove.insert(dist(gen));
      //copy indices to array and copy to GPU
      T *removeArray = new T[num_indices - k];
      int idx = 0;
      for(auto it = remove.begin(); it != remove.end(); it++)
        removeArray[idx++] = *it;
      T *device_removeArray;
      hipMalloc((void**)&device_removeArray, sizeof(T) * (num_indices-k));
      hipMemcpyAsync(device_removeArray, removeArray, sizeof(T) * (num_indices-k), hipMemcpyHostToDevice);

      markRemove<<<(num_indices-k)/num_threads_per_group*4, num_threads_per_group*4>>>(device_data, num_indices, device_removeArray, num_indices-k);
      #if 1
      //compaction
      int removed = thrust::copy_if(thrust::device, device_data, device_data+num_indices, device_data, less_functor<T>(universe)) - device_data;
      #else
      //copy elements and count collisions, restart until no collisions left
      remove_remaining = num_indices-k;
      while(remove_remaining > 0)
      {
        replace<<<remove_remaining, num_threads_per_group>>>(device_data, k+remove_remaining, device_removeArray, num_indices-k);
        remove_remaining = thrust::copy_if(thrust::device, device_removeArray, device_removeArray + remove_remaining, device_removeArray, less_functor<T>((T)universe))-device_removeArray; 
      }

      hipFree(&device_removeArray);
      #endif
      hipFree(&device_removeArray);
      delete [] removeArray;
      //read back data
      //hipMemcpy(&dest[0], device_data, n*sizeof(T), hipMemcpyDeviceToHost);
    }
};
template <typename T> cuda_generator<T> *cuda_generator<T>::_instance = 0;

struct cuda_gen {
  template <typename It>
    static void generate_block(It dest, size_t size, size_t k, size_t universe, double p,
        unsigned long long seed = 0)
    {
      using value_type = typename std::iterator_traits<It>::value_type;
      cuda_generator<value_type> *generator = cuda_generator<value_type>::instance();

      generator->generate_block(dest, size, k, universe, p, seed);
    }

  template <typename It>
    static void generate_block(It begin, It end, double p,
        unsigned int long long = 0)
    {
      generate_block(begin, end-begin, p, seed);
    }
};

// copied from include/sampler.h
// Formulas from "Sequential Random Sampling" by Ahrens and Dieter, 1985
static std::pair<double, size_t> calc_params(size_t universe, size_t k /* samples */) {
  double r = sqrt(k);
  double a = sqrt(log(1+k/(2*M_PI)));
  a = a + a*a/(3.0 * r);
  size_t b = k + size_t(4 * a * r);
  double p = (k + a * r) / universe;
  return std::make_pair(p, b);
}

int main(int argc, char **argv)
{
  hipSetDevice(3);
  size_t num_threads = 1;
  arg_parser args(argc, argv);
  size_t universe = args.get<size_t>("n", 1<<30);
  size_t k = args.get<size_t>("k", 1<<20); // sample size

  size_t iterations = args.get<size_t>("i", (1<<30)/k);
  const bool verbose = args.is_set("v") || args.is_set("vv");
  const bool very_verbose = args.is_set("vv");
  const bool quiet = args.is_set("q");

  double p; size_t ssize;
  std::tie(p, ssize) = calc_params(universe, k);
  #ifdef USE64BIT
  std::vector<unsigned long long> vec(ssize);
  #else
  std::vector<unsigned int> vec(ssize);
  #endif
  statistics mt_stats;

  // warmup
  // k = number of indices (output), p = probability, universe = maximum index size
  // ssize = size of index array
  cuda_gen::generate_block(vec.begin(), ssize, k, universe, p);

  std::stringstream extra_stream;
  extra_stream << " k=" << k << " b=" << ssize
    << " p=" << p << " N=" << universe;
  auto extra = extra_stream.str();

  // Measure
  hipDeviceSynchronize();

  // stats for multi-threaded version including sync / load imbalance
  timer t;
  for(int i = 0; i < iterations; i++)
  {
    cuda_gen::generate_block(vec.begin(), ssize, k, universe, p);
    hipDeviceSynchronize();
    double time = t.get_and_reset();
    mt_stats.push(time);
  }
  std::cout << " mt_time=" << mt_stats.avg()
    << " mt_dev=" << mt_stats.stddev()
    << " numthreads=" << num_threads
    << " iterations=" << iterations
    << extra << std::endl;
  #ifdef PROFILING
  hipProfilerStop();
  #endif
}
