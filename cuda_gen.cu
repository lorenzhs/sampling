#include "hip/hip_runtime.h"
#include <cassert>
#include <limits>
#include <random>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#ifdef PROFILING
#include <hip/hip_runtime_api.h>
#endif
#include "util.h"
#include "timer.h"
#include <stocc/stocc.h>
#include <randomc/randomc.h>

#include <thrust/copy.h>
#include <thrust/find.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

#include <include/arg_parser.h>

#include <algorithm>
#include <cmath>
#include <utility>

#ifndef M_PI
#define M_PI 3.14159265358979323814
#endif

//initialize hiprand states
__global__ void initCurand(hiprandState *state, unsigned long long *seed)
{
  size_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(seed[threadId], 0, 0, &state[threadId]);
}

//sample random numbers with geometric distribution
  template <typename T>
__global__ void sample(hiprandState *state, T *data, const size_t count, const T num, const double oneminusp)
{
  size_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
  size_t idx = threadId * num;
  hiprandState mState = state[threadId];
  //each thread generates 'num' random numbers with geometric distribution
  float lpinv = 1/log2(oneminusp);
  for(unsigned int i = 0; i < num && idx < count; i+=4, idx+=4)
  {
    float4 r = make_float4(__log2f(1-hiprand_uniform(&mState)),__log2f(1-hiprand_uniform(&mState)),__log2f(1-hiprand_uniform(&mState)),__log2f(1-hiprand_uniform(&mState)));
    if(i < num && idx < count)
      data[idx] = r.x*lpinv+1;
    if(i+1 < num && idx+1 < count)
      data[idx+1] = r.y*lpinv+1;
    if(i+2 < num && idx+2 < count)
      data[idx+2] = r.z*lpinv+1;
    if(i+3 < num && idx+3 < count)
      data[idx+3] = r.w*lpinv+1;
  }
  state[threadId] = mState;
}

#if 0
//naive way of removing excessive indices
  template <typename T>
__global__ void naive(hiprandState *state, T *data, const T *split, const T *to_remove)
{
  size_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
  T idx = split[threadId];
  T num = split[threadId+1]-split[threadId];
  hiprandState mState = state[threadId];

  for(unsigned int i = 0; i < to_remove[threadId]; i++)
  {
    //pick a random element and remove it if it wasn't removed before
    T r = hiprand_uniform(&mState)*num;
    if(data[idx+r] == (T)-1)
      --i;
    data[idx+r] = (T)-1;
  }
  state[threadId] = mState;
}
#endif

//Vitter's Algorithm A (base case of Algorithm D)
  template <typename T>
__global__ void algorithmA(hiprandState *state, T *data, const T *split, const T *to_remove)
{
  size_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
  T idx = split[threadId];
  T num = split[threadId+1]-split[threadId];
  hiprandState mState = state[threadId];

  float top = num - to_remove[threadId];
  float Nreal = num;
  float r = hiprand_uniform(&mState);  
  float quot = top/Nreal;

  for(unsigned int i = 0; i < num; i++, idx++)
  {
    if(quot <= r)
    {
      r = hiprand_uniform(&mState);
      data[idx+1] = (T)-1;
      quot = 1;
    } else
      top--;
    Nreal--;
    quot *= top/Nreal;
  }
  state[threadId] = mState;
}

//functor for filtering elements less or equal a constant
template <typename T>
struct less_equal_functor
{
  T max_idx;
  less_equal_functor(const T _max_idx)
    : max_idx(_max_idx)
  { }

  __host__ __device__
    bool operator()(T x)
    {
      return x <= max_idx;
    }
};

#if 0
//functor for filtering elements greater than a constant
template <typename T>
struct greater_than_functor
{
  T max_idx;
  greater_than_functor(const T _max_idx)
    : max_idx(_max_idx)
  { }

  __host__ __device__
    bool operator()(T x)
    {
      return x > max_idx;
    }
};
#endif

//initialize prng only once (seed can be set nevertheless)
template <typename T>
class cuda_generator
{
  private:
    static cuda_generator<T> *_instance;

    T *device_data;

    //(GPU-)states for random number generators
    hiprandState *state;
    //seeds on CPU and GPU for each thread of the random number generator
    unsigned long long *seeds, *seeds_device;
    size_t N;
    const T num_threads = 1<<16;
    const T num_threads_per_group = 1<<3;
    const T num_groups = num_threads / num_threads_per_group;
    T *device_split, *device_to_remove, *split, *to_remove;

    cuda_generator()
    {
      N = 0;
      hipMalloc((void**)&state, num_threads * sizeof(hiprandState));
      hipMalloc((void**)&seeds_device, num_threads * sizeof(unsigned long long));
      seeds = new unsigned long long[num_threads];
      device_data = 0;
      split = new T[num_threads+1];
      to_remove = new T[num_threads];

      //remove excessive elements per processor
      hipMalloc((void**)&device_split, sizeof(T)*(num_threads+1));
      hipMalloc((void**)&device_to_remove, sizeof(T)*num_threads);
    }

    ~cuda_generator()
    {
      if(split)
        delete [] split;
      if(to_remove)
        delete [] to_remove;
      hipFree((void**)&device_split);
      hipFree((void**)&device_to_remove);
      hipFree((void**)&state);
      hipFree((void**)&seeds_device);
    }


  public:
    static cuda_generator<T> *instance()
    {
      if(!_instance)
        _instance = new cuda_generator<T>();
      return _instance;
    }

    static void sampleR(T n, T N, T j, T k, T *startIdx, T *numSamples, StochasticLib1 stoc = StochasticLib1(0), T off = 0)
    {
      if(k == j+1)
      {
        numSamples[j] = n;
        startIdx[j] = N+off;
        return;
      }
      T N2 = N/2;
      T x = stoc.Hypergeometric(n, N2, N);
      sampleR(x, N2, j, (j+k)/2, startIdx, numSamples, stoc, off);
      sampleR(n-x, N-N2, (j+k)/2, k, startIdx, numSamples, stoc, off+N2);
    }

    template <typename It>
      void generate_block(It dest, size_t size, size_t k, size_t universe, double p,
          unsigned long long seed = 0)
      {
        //fprintf(stderr, "%d: universe=%Iu\n", __LINE__, universe);
        using value_type = typename std::iterator_traits<It>::value_type;
        assert(p > 0 && p < 1);

        if (seed == 0)
        {
          seed = std::random_device{}();
        }

        //allocate memory on GPU if blocksize changed
        if(N != size)
        {
          if(device_data)
            hipFree(device_data);
          hipMalloc((void**)&device_data, sizeof(T)*size);
          N = size;
        }
        for(int i = 0; i < num_threads; i++)
          seeds[i] = seed+i;
        hipMemcpy(seeds_device, seeds, num_threads*sizeof(unsigned long long), hipMemcpyHostToDevice);
        //initialize prng-states
        initCurand<<<num_groups, num_threads_per_group>>>(state, seeds_device);
        //fill array with geometrically distributed random numbers
        T samples_per_thread = max((T)ceil(N/(double)num_threads), (T)1);
        T num_indices = 0;
        timer t;
        do {
          sample<<<num_groups, num_threads_per_group>>>(state, device_data, N, samples_per_thread, 1.0-p);
          //prefix-sum for calculating array-indices
          thrust::inclusive_scan(thrust::device, device_data, device_data+N, device_data);
          //count_if is faster than the other two.
          num_indices = thrust::count_if(thrust::device, device_data, device_data+N, less_equal_functor<T>((T)universe));
          //num_indices = thrust::find_if(thrust::device, device_data, device_data+N, less_equal_functor<T>((T)universe)) - device_data;
          //num_indices = thrust::copy_if(thrust::device, device_data, device_data+N, device_data, less_equal_functor<T>((T)universe)) - device_data;
        } while(num_indices < k);
        hipDeviceSynchronize();

        //divide array using hypergeometric deviates (one subarray per processor)
        sampleR(num_indices-k, num_indices, 0, num_threads, split+1, to_remove);
        split[0] = 0;
        //copy start- and end-indices of subarrays as well as the number
        //of elements to be sampled from these subarrays to GPU memory
        hipMemcpy(device_split, split, sizeof(T)*(num_threads+1), hipMemcpyHostToDevice);
        hipMemcpy(device_to_remove, to_remove, sizeof(T)*num_threads, hipMemcpyHostToDevice);

        //sample elements (elements are removed by setting the index to some value larger
        //than the universe size
        //naive<<<num_groups, num_threads_per_group>>>(state, device_data, device_split, device_to_remove);
        algorithmA<<<num_groups/4, num_threads_per_group*4>>>(state, device_data, device_split, device_to_remove);

        //hipDeviceSynchronize();

        //compaction
        thrust::copy_if(thrust::device, device_data, device_data+num_indices, device_data, less_equal_functor<T>(universe));
        //read back data
        //hipMemcpy(&dest[0], device_data, n*sizeof(T), hipMemcpyDeviceToHost);
      }
};
template <typename T> cuda_generator<T> *cuda_generator<T>::_instance = 0;

struct cuda_gen {
  template <typename It>
    static void generate_block(It dest, size_t size, size_t k, size_t universe, double p,
        unsigned long long seed = 0)
    {
      using value_type = typename std::iterator_traits<It>::value_type;
      cuda_generator<value_type> *generator = cuda_generator<value_type>::instance();

      generator->generate_block(dest, size, k, universe, p, seed);
    }

  template <typename It>
    static void generate_block(It begin, It end, double p,
        unsigned int long long = 0)
    {
      generate_block(begin, end-begin, p, seed);
    }
};

// copied from include/sampler.h
// Formulas from "Sequential Random Sampling" by Ahrens and Dieter, 1985
static std::pair<double, size_t> calc_params(size_t universe, size_t k /* samples */) {
  double r = sqrt(k);
  double a = sqrt(log(1+k/(2*M_PI)));
  a = a + a*a/(3.0 * r);
  size_t b = k + size_t(4 * a * r);
  double p = (k + a * r) / universe;
  return std::make_pair(p, b);
}

int main(int argc, char **argv)
{
  hipSetDevice(1);
  size_t num_threads = 1;
  arg_parser args(argc, argv);
  size_t universe = args.get<size_t>("n", 1<<30);
  size_t k = args.get<size_t>("k", 1<<20); // sample size

  size_t iterations = args.get<size_t>("i", (1<<30)/k);
  const bool verbose = args.is_set("v") || args.is_set("vv");
  const bool very_verbose = args.is_set("vv");
  const bool quiet = args.is_set("q");

  double p; size_t ssize;
  std::tie(p, ssize) = calc_params(universe, k);
  #ifdef USE64BIT
  std::vector<unsigned long long> vec(ssize);
  #else
  std::vector<unsigned int> vec(ssize);
  #endif
  int version = 0;
  hiprandGetVersion(&version);
  fprintf(stderr, "%d", version);
  statistics mt_stats;

  // warmup
  // k = number of indices (output), p = probability, universe = maximum index size
  // ssize = size of index array
  cuda_gen::generate_block(vec.begin(), ssize, k, universe, p);
  //std::cout << "Running warmup (" << k << " samples)" << std::endl;

  std::stringstream extra_stream;
  extra_stream << " k=" << k << " b=" << ssize
    << " p=" << p << " N=" << universe;
  auto extra = extra_stream.str();

  //std::cout << "Running measurements..." << std::endl;

  // Measure
  hipDeviceSynchronize();

  // stats for multi-threaded version including sync / load imbalance
  timer t;
  for(int i = 0; i < iterations; i++)
  {
    cuda_gen::generate_block(vec.begin(), ssize, k, universe, p);
    hipDeviceSynchronize();
    double time = t.get_and_reset();
    mt_stats.push(time);
  }
  std::cout << " mt_time=" << mt_stats.avg()
    << " mt_dev=" << mt_stats.stddev()
    << " numthreads=" << num_threads
    << " iterations=" << iterations
    << extra << std::endl;
  #ifdef PROFILING
  hipProfilerStop();
  #endif
}
