#include "hip/hip_runtime.h"
#include <cassert>
#include <limits>
#include <random>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h>
#include <chrono>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

//initialize hiprand states
__global__ void initCurand(hiprandState *state, unsigned long long *seed)
{
  unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(seed[threadId], 0, 0, &state[threadId]);
}

//sample random numbers with geometric distribution
template <typename T>
__global__ void sample(hiprandState *state, T *data, const T num, const double oneminusp)
{
  unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int idx = threadId * num;
  hiprandState mState = state[threadId];
  //each thread generates 'num' random numbers with geometric distribution
  for(unsigned int i = 0; i < num; i++, idx++)
    data[idx] = __log2f(1.0f-hiprand_uniform(&mState))/__log2f(oneminusp)+1;
  state[threadId] = mState;
}

//naive way of removing excessive indices
template <typename T>
__global__ void naive(hiprandState *state, T *data, const T *split, const T *to_remove)
{
  unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int idx = split[threadId];
  unsigned int num = split[threadId+1]-split[threadId];
  hiprandState mState = state[threadId];

  for(unsigned int i = 0; i < to_remove[threadId]; i++)
  {
    //pick a random element and remove it if it wasn't removed before
    unsigned int r = hiprand_uniform(&mState)*num;
    if(data[idx+r] == (T)-1)
      --i;
    data[idx+r] = (T)-1;
  }
  state[threadId] = mState;
}

//Vitter's Algorithm A (base case of Algorithm D)
template <typename T>
__global__ void algorithmA(hiprandState *state, T *data, const T *split, const T *to_remove)
{
  unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int idx = split[threadId];
  unsigned int num = split[threadId+1]-split[threadId];
  hiprandState mState = state[threadId];

  float top = num - to_remove[threadId];
  float Nreal = num;
  float r = hiprand_uniform(&mState);  
  float quot = top/Nreal;

  for(unsigned int i = 0; i < num; i++, idx++)
  {
    if(quot <= r)
    {
      r = hiprand_uniform(&mState);
      data[idx+1] = (T)-1;
      quot = 1;
    } else
      top--;
    Nreal--;
    quot *= top/Nreal;
  }
  state[threadId] = mState;
}

//functor for filtering elements less or equal a constant
template <typename T>
struct less_equal_functor
{

  T max_idx;
  less_equal_functor(const T _max_idx)
    : max_idx(_max_idx)
  { }

  __host__ __device__
    bool operator()(T x)
    {
      return x <= max_idx;
    }
};

//functor for filtering elements greater than a constant
template <typename T>
struct greater_than_functor
{
  T max_idx;
  greater_than_functor(const T _max_idx)
    : max_idx(_max_idx)
  { }

  __host__ __device__
    bool operator()(T x)
    {
      return x > max_idx;
    }
};

//initialize prng only once (seed can be set nevertheless)
template <typename T>
class cuda_generator
{
  private:
    static cuda_generator<T> *_instance;

    T *device_data;

    //(GPU-)states for random number generators
    hiprandState *state;
    //seeds on CPU and GPU for each thread of the random number generator
    unsigned long long *seeds, *seeds_device;
    //
    size_t N;
    const T num_threads = 1<<10;
    const T num_threads_per_group = 1<<4;
    const T num_groups = num_threads / num_threads_per_group;

    cuda_generator()
    {
      N = 0;
      hipMalloc((void**)&state, num_threads * sizeof(hiprandState));
      hipMalloc((void**)&seeds_device, num_threads * sizeof(unsigned long long));
      seeds = new unsigned long long[num_threads];
      device_data = 0;
    }


  public:
    static cuda_generator<T> *instance()
    {
      if(!_instance)
        _instance = new cuda_generator<T>();
      return _instance;
    }
	
	//first call: num_samples, N, 0, num_threads, ...
	static void sampleR(T n, T N, T j, T k, T *startIdx, T *numSamples)
	{
		if(k == j+1)
		{
			numSamples[j] = n;
			startIdx[j] = N;
			return;
		}
		T N2 = N/2;
		T x = n/2;//StochasticLib1::Hypergeometric(n, N2, N);
		sampleR(x, N2, j, (j+k)/2, startIdx, numSamples);
		sampleR(n-x, N-N2, (j+k)/2, k, startIdx, numSamples);
		for(int i = (j+k)/2; i < k; i++)
		  startIdx[i] += N2;
	}

    template <typename It>
      void generate_block(It dest, size_t size, double p,
          unsigned long long seed = 0)
      {
        using value_type = typename std::iterator_traits<It>::value_type;
        assert(p > 0 && p < 1);

        if (seed == 0)
        {
          seed = std::random_device{}();
        }

        //allocate memory on GPU if blocksize changed
        if(N != size)
        {
          if(device_data)
            hipFree(device_data);
          hipMalloc((void**)&device_data, sizeof(T)*size);
          N = size;
        }
        for(int i = 0; i < num_threads; i++)
          seeds[i] = seed+i;
        hipMemcpy(seeds_device, seeds, num_threads*sizeof(unsigned long long), hipMemcpyHostToDevice);
        //initialize prng-states
        initCurand<<<num_groups, num_threads_per_group>>>(state, seeds_device);
        //fill array with geometrically distributed random numbers
        sample<<<num_groups, num_threads_per_group>>>(state, device_data, (T)(N/num_threads), 1.0f-p);

        //prefix-sum for calculating array-indices
        thrust::inclusive_scan(thrust::device, device_data, device_data+N, device_data);

        //TODO: get universe size from command line parameter
        T universe = 1.5*size;

        //count number of indices that are small enough
        T num_indices = thrust::count_if(thrust::device, device_data, device_data+N, less_equal_functor<T>(universe));

        //divide array using hypergeometric deviates (one subarray per processor)
        T n = 1<<28;
		#if 0
        //TODO!!! - for now just take subarrays of equal size
        //intervals for each thread:
        T *split = new T[num_threads+1];
        T *to_remove = new T[num_threads];
        T remove_remaining = num_indices-n;
        T remove_total = 0;
        for(int i = 0; i < num_threads+1; i++)
        {
          split[i] = (double)i*n/(double)num_threads+0.5;
          unsigned int thread_removes = (remove_remaining)/(double)(num_threads-i);
          remove_total += thread_removes;
          to_remove[i] = thread_removes;
          remove_remaining -= thread_removes;
        }
		#else
        T *split = new T[num_threads+1];
        T *to_remove = new T[num_threads];
		
		sampleR(n, N, 0, num_threads, split+1, to_remove);
		split[0] = 0;
		/*
		for(int i = 0; i <= num_threads; i++)
			fprintf(stderr, "%d ", split[i]);
		for(int i = 0; i < num_threads; i++)
			fprintf(stderr, "%d ", to_remove[i]);
		*/
		#endif

        //remove excessive elements per processor
        T *device_split, *device_to_remove;
        //copy start- and end-indices of subarrays as well as the number
        //of elements to be sampled from these subarrays to GPU memory
        hipMalloc((void**)&device_split, sizeof(T)*(num_threads+1));
        hipMalloc((void**)&device_to_remove, sizeof(T)*num_threads);
        hipMemcpy(device_split, split, sizeof(T)*(num_threads+1), hipMemcpyHostToDevice);
        hipMemcpy(device_to_remove, to_remove, sizeof(T)*num_threads, hipMemcpyHostToDevice);

        //sample elements (elements are removed by setting the index to some value larger
        //than the universe size
        //naive<<<num_groups, num_threads_per_group>>>(state, device_data, device_split, device_to_remove);
        algorithmA<<<num_groups, num_threads_per_group>>>(state, device_data, device_split, device_to_remove);

        //compaction
        T final_count = thrust::copy_if(thrust::device, device_data, device_data+num_indices, device_data, less_equal_functor<T>(universe)) - device_data;
		//T final_count = thrust::remove_copy(thrust::device, device_data, device_data+num_indices, device_data, (T)-1) - device_data;
		fprintf(stderr, "result: %lu / %lu elements\n", final_count, n);
        //read back data
        //hipMemcpy(&dest[0], device_data, N*sizeof(T), hipMemcpyDeviceToHost);
      }
};
template <typename T> cuda_generator<T> *cuda_generator<T>::_instance = 0;

struct cuda_gen {
  template <typename It>
  static void generate_block(It dest, size_t size, double p,
      unsigned int seed = 0)
  {
    using value_type = typename std::iterator_traits<It>::value_type;
    cuda_generator<value_type> *generator = cuda_generator<value_type>::instance();
    generator->generate_block(dest, size, p, seed);
  }

  template <typename It>
  static void generate_block(It begin, It end, double p,
      unsigned int seed = 0)
  {
    generate_block(begin, end-begin, p, seed);
  }
};

int main(int argc, char **argv)
{
  unsigned int N = 1024*1024*512;
  std::vector<unsigned int> vec(N);
  fprintf(stderr, "\n");
  hipSetDevice(3);
  do 
  {
    //unsigned int num_threads = 1;
        cuda_gen::generate_block(vec.begin(), N, 0.5);
    std::chrono::time_point<std::chrono::high_resolution_clock> t = std::chrono::high_resolution_clock::now();
    //#pragma omp parallel num_threads(1)
    {
      //cuda_generator<unsigned int> *generator = new cuda_generator<unsigned int>(N/num_threads);
      hipDeviceSynchronize();
      hipProfilerStart();
      for(int i = 0; i < 10; i++)
        //generator->generate_block(vec.begin()/*+omp_get_thread_num()*N/num_threads*/, N/num_threads, 0.5);
        cuda_gen::generate_block(vec.begin(), N, 0.5);
      hipProfilerStop();
      hipDeviceSynchronize();
    }
    //N /= 2;
    float dt = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - t).count();
    fprintf(stderr, "%d %f\n", N, dt/10);
  } while(N > N);
  //fprintf(stderr, "done\n");
  double avg = 0;
  avg = vec[N-1]/(double)N;
  fprintf(stderr, "avg: %f\n", avg);
  for(int i = 0; i < 100; i++)
  {
	fprintf(stderr, "%lu ", vec[i]);
  }
  return 0;
}
