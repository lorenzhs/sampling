#include "hip/hip_runtime.h"
#pragma once

#include <cassert>
#include <limits>
#include <random>
#include <hiprand.h>
#include <hip/hip_runtime_api.h> //For profiling
#include <chrono> //For profiling
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/system/cuda/execution_policy.h>
#include <omp.h>
#include <map>


// convert a linear index to a row index
struct row_index : public thrust::unary_function<size_t,size_t>
{
	size_t c; // number of columns
	__host__ __device__
	row_index(size_t _c)
      : c(_c) 
	{ }

  __host__ __device__
  size_t operator()(size_t i)
  {
    return i / c;
  }
};

//Sample number with geometric distribution:
template <typename T>
struct geometric_distribution : public thrust::unary_function<T,T>
{
  #define DOUBLES
  #ifdef DOUBLES
  const double p;
  const double maxvalinv;
  #else
  const float p;
  const float maxvalinv;
  #endif

  geometric_distribution(const double _p, const double _maxvalinv)
    : p(_p),
    maxvalinv(_maxvalinv)
  { }

  __device__
  T operator()(T x)
  {
    return __log2f(1.0f-maxvalinv*x)/__log2f(1.0f-p)+1;
    //return log1p(-_maxvalinv*(double)x)/log1p(-_p);
  }
};

//initialize prng only once (seed can be set nevertheless)
template <typename T>
class cuda_generator
{
  private:
    static cuda_generator<T> *_instance;
	//static std::map<int, cuda_generator<T>*> _instance;
    thrust::device_vector<T> device_data;
	//T *output;
    size_t N;
    hiprandGenerator_t prng;
	hipStream_t s;
  public:
    cuda_generator(size_t size)
    {
  fprintf(stderr, "creating instance\n");
  fflush(stderr);
	  hipSetDevice(omp_get_thread_num());
      N = 0;
      device_data = thrust::device_vector<T>(size);
      N = size;
      //hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
      //hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_MTGP32);
      hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_MT19937);
	  //output = 0;
	  hipStreamCreate(&s);
	  hiprandSetStream(prng, s);
    }

    void call_curand_generate()
    {
      hiprandGenerate(prng, thrust::raw_pointer_cast(device_data.data()), N);
    }

  //public:
  static cuda_generator<T> *instance()
  {
	  int tid = omp_get_thread_num();
  fprintf(stderr, "returning instance %d\n", tid);
  fflush(stderr);
  //return new cuda_generator<T>();
  //#pragma omp critical
    if(_instance)
      _instance = new cuda_generator<T>();
    return _instance;
  }

  template <typename It>
  void generate_block(It dest, size_t size, double p,
                      unsigned long long seed = 0)
  {
    using value_type = typename std::iterator_traits<It>::value_type;			
    assert(p > 0 && p < 1);

    if (seed == 0) {
      seed = std::random_device{}();
    }

    //allocate memory on GPU if blocksize changed
    if(N != size)
    {
	  fprintf(stderr, "reallocating (%d --> %d)\n", N, size);
      device_data = thrust::device_vector<T>(size);
      N = size;
	  /*
	  if(output)
		  hipFree(output);
	  hipMalloc(&output, N*sizeof(T));
	  */
    }

    //set seed of prng
    hiprandSetPseudoRandomGeneratorSeed(prng, seed);

    //fill array with random numbers (should depend on size of the data type)
    call_curand_generate();

    //fprintf(stderr, ".");

    //geometric distribution and prefix sum
    auto tbegin = thrust::make_transform_iterator(device_data.begin(), geometric_distribution<T>(p, 1.0/(std::numeric_limits<T>::max())));
    auto tend = thrust::make_transform_iterator(device_data.end(), geometric_distribution<T>(p, 1.0/(std::numeric_limits<T>::max())));
	thrust::transform(device_data.begin(), device_data.end(), device_data.begin(), geometric_distribution<T>(p, 1.0/(std::numeric_limits<T>::max())));
	#if 0
	//thrust::inclusive_scan(thrust::system::cuda::par, tbegin, tend, device_data.begin());// Determine temporary device storage requirements for inclusive prefix sum
	void     *d_temp_storage = NULL;
	size_t   temp_storage_bytes = 0;
	hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, &device_data[0], output, N,0,1);
	// Allocate temporary storage for inclusive prefix sum
	hipMalloc(&d_temp_storage, temp_storage_bytes);
	// Run inclusive prefix sum
	hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, &device_data[0], output, N,0,1);
	hipFree(d_temp_storage);
	#else
	#if 0
    {
	  //thrust::inclusive_scan(thrust::system::cuda::par, device_data.begin(), device_data.end(), output);
      thrust::counting_iterator<size_t> indices(0);
      //auto tbegin = thrust::make_transform_iterator(indices, row_index<size_t>(N/2));
      //auto tend = thrust::make_transform_iterator(indices, row_index<size_t>(N/2))+N*sizeof(T);
      //thrust::inclusive_scan_by_key(tbegin, tend, device_data.begin(), device_data.begin());
	  for(int n = N; n >= 1; n >>= 1)
	  {
		  /*
		  thrust::inclusive_scan_by_key(thrust::make_transform_iterator(indices, row_index(N/2)),
			thrust::make_transform_iterator(indices, row_index(N/2)) + N,
			device_data.begin(),
			device_data.begin());
		  */
		  auto row_iterator = thrust::make_transform_iterator(indices, row_index(n));
		  std::chrono::time_point<std::chrono::high_resolution_clock> t = std::chrono::high_resolution_clock::now();
		  for(int i = 0; i < 100; i++)
			  thrust::inclusive_scan_by_key(row_iterator,
				row_iterator + N,
				device_data.begin(),
				device_data.begin());
		  float dt = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - t).count();
		  fprintf(stderr, "%d %f\n", n, dt/100);
	  }
    }
	#else
    {
	  thrust::inclusive_scan(thrust::system::cuda::par.on(s), device_data.begin(), device_data.end(), device_data.begin());
    }
	#endif
	#endif
	
    //read back data
    //hipMemcpy(&dest[0], device_data.data().get(), N*sizeof(T), hipMemcpyDeviceToHost);
	//hipMemcpy(&dest[0], output, N*sizeof(T), hipMemcpyDeviceToHost);
	//hipDeviceSynchronize();
  }
};
//template <typename T> std::map<int, cuda_generator<T>*> cuda_generator<T>::_instance;
template <typename T> cuda_generator<T>* cuda_generator<T>::_instance;
/*
//TODO: not working!!! hiprand complains about not using a 64-bit prng.
template <>
void cuda_generator<unsigned long long>::call_curand_generate()
{
  hiprandGenerateLongLong(prng, thrust::raw_pointer_cast(device_data.data()), N);
}
*/

struct cuda_gen {
  template <typename It>
  static void generate_block(It dest, size_t size, double p,
                             unsigned int seed = 0)
  {
    using value_type = typename std::iterator_traits<It>::value_type;
    cuda_generator<value_type> *generator = cuda_generator<value_type>::instance();
    generator->generate_block(dest, size, p, seed);
  }

  template <typename It>
  static void generate_block(It begin, It end, double p,
                             unsigned int seed = 0)
  {
    generate_block(begin, end-begin, p, seed);
  }
};

#if 1
//For profiling
int main(int argc, char **argv)
{
  unsigned int N = 1024*1024*512;
  std::vector<unsigned int> vec(N);
  fprintf(stderr, "\n");
  //hipSetDevice(1);
  do {
	  int num_threads = 1;
	  std::chrono::time_point<std::chrono::high_resolution_clock> t = std::chrono::high_resolution_clock::now();
	  #pragma omp parallel num_threads(1)
	  {
		  cuda_generator<unsigned int> *generator = new cuda_generator<unsigned int>(N/num_threads);
		  hipDeviceSynchronize();
  hipProfilerStart();
	  for(int i = 0; i < 10; i++)
		generator->generate_block(vec.begin()+omp_get_thread_num()*N/num_threads, N/num_threads, 0.5);
  hipProfilerStop();
	  }
	  //N /= 2;
	  float dt = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - t).count();
	  fprintf(stderr, "%d %f\n", N, dt/10);
  } while(N > N);
  //fprintf(stderr, "done\n");
  double avg = 0;
  avg = vec[N-1]/(double)N;
  fprintf(stderr, "avg: %f\n", avg);
  return 0;
}
#endif
